#include "hip/hip_runtime.h"
#include "util.h"
#define TPB 1024
#define NB 2147483647


__global__ void testgpu(char *text, char *pattern, int *match, int pattern_size, int text_size)
{
    printf("%d> %s, %s, %d, %d, %d\n", threadIdx.x + blockIdx.x*blockDim.x, text, pattern, match[threadIdx.x + blockIdx.x*blockDim.x], pattern_size, text_size);
}

__global__ void match(char *text, char *pattern, int *match, int pattern_size, int text_size){

    int pid = threadIdx.x + blockIdx.x*blockDim.x;

    if (pid <= text_size - pattern_size){
    
        int flag = 1; 
        for (int i = 0; i < pattern_size; i++){
            if (text[pid+i] != pattern[i]){
                    flag = 0;
            }
        }
        match[pid] = flag;
    }
    // printf("\t\tpid=%d, %s in %s: %d\n", pid, pattern, text, match[pid]);
    return;
}


void GPUinfo()
{
    int num_gpus;
    size_t free, total;
    hipGetDeviceCount( &num_gpus );
    for ( int gpu_id = 0; gpu_id < num_gpus; gpu_id++ ) {
        hipSetDevice( gpu_id );
        int id;
        hipGetDevice( &id );
        hipMemGetInfo( &free, &total );
        // cout << "GPU " << id << " memory: free=" << free << ", total=" << total << endl;
        printf("GPU %d free=%ld | total=%ld\n", id, free, total);
    }
    return;

}

int main(int argc, char **argv)
{
    if (argc != 5)
    {
        printf("Wrong argments usage: ./kmer [REFERENCE_FILE] [READ_FILE] [k] [OUTPUT_FILE]\n");
    }

    FILE *fp;
    int kmer;

    char *reference_str = (char *)malloc(MAX_REF_LENGTH * sizeof(char));
    char *read_str = (char *)malloc(MAX_READ_LENGTH * sizeof(char));

	char *reference_filename;
	char *read_filename;
	char *output_filename;
    int reference_length;

    reference_filename = argv[1];
    read_filename = argv[2];
    kmer = atoi(argv[3]);
    output_filename = argv[4];


    fp = fopen(reference_filename, "r");

    if (fgets(reference_str, MAX_REF_LENGTH, fp) == NULL)
    { //A single line only
        printf("Problem in file format!\n");
        return 1;
    }

    substring(reference_str, 0, strlen(reference_str) - 1);
    // printf("Reference str is = %s\n", reference_str);
    fclose(fp);

    //Read queries
    StringList queries;
	initStringList(&queries, 3); // initially 3 reads
	

    int success = read_file(read_filename, &queries);
	// char buff[kmer];
    // buff[kmer] = '\0';
    // int matches[queries.used];
	reference_length = strlen(reference_str); //Last character is '\n'
	

    int h_matches[reference_length] = {0};
    int* d_matches = NULL;
    char* d_reference = NULL;
    char* d_kmer;
    hipMalloc((void **)&d_reference, reference_length*sizeof(char));
    hipMalloc((void **)&d_matches, reference_length*sizeof(int));
    hipMalloc((void **)&d_kmer, kmer*sizeof(char));
    hipMemcpy(d_reference, reference_str, reference_length*sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_matches, h_matches, reference_length*sizeof(int), hipMemcpyHostToDevice);


    char buff[kmer];
    // buff[kmer] = '\0';
    int matches[queries.used] = {0};
    int sum =0;


	FILE* out_f = fopen(output_filename, "w");


    
        int num_blocks = reference_length / TPB +1;
        
        for (int u = 0; u < queries.used; u++)
        {
            // printf("read %d: %s\n", u, queries.array[u]);
            
            int k = 0;
            bool end = 0;
            while (!end)
            {
                substring(buff, queries.array[u], k, k+kmer);
                hipMemcpy(d_kmer, buff, kmer*sizeof(char), hipMemcpyHostToDevice);
                
                // matches[u] += KMPSearch(buff, kmer, reference_str, reference_length);

                
                // printf("\n\tkmer %d: %s\n", k, buff);
                match<<<num_blocks, TPB>>>(d_reference, d_kmer, d_matches, kmer, reference_length);
                

                hipError_t cudaerr = hipDeviceSynchronize();
                if (cudaerr != hipSuccess)
                    printf("kernel launch failed with error \"%s\".\n", hipGetErrorString(cudaerr));
                // else
                    // printf("no errors in kernel launch\n");

                hipMemcpy(h_matches, d_matches, reference_length*sizeof(int), hipMemcpyDeviceToHost);


                sum = 0;
                #pragma omp parallel for shared(sum, h_matches) reduction(+: sum)
                for (int i = 0; i < reference_length; i++)
                {
                    // printf("h_matches[%d]=%d\n", i, h_matches[i]);
                    sum += h_matches[i];
                }
                matches[u] += sum;

                if (queries.array[u][k+kmer+1]=='\0')
                    end = 1;
                else
                    k++;
            }
            // printf("matches for read %d: %d\n",u, matches[u]);
        }


	// printf("\n");

	// fprintf(out_f, "bai\n");
	for(int r = 0; r < queries.used; r++)
	{
		fprintf(out_f, "%d\n", matches[r]);
    }
    

	fclose(out_f);
    //Free up
    freeStringList(&queries);

    free(reference_str);
    free(read_str);
}
